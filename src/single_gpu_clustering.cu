/* 
 *
 * Docs
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <float.h>
#include <math.h>

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

/* Config params */
#define PRINT_LOG 1
#define PRINT_ANALYSIS 1
/* Define constants */
#define RANGE 100

/**************************** Definitions *************************************/

// Function declarations
void  seq_clustering(float *, unsigned int, unsigned int, int *, float *);
void  gpu_clustering(float *, unsigned int, unsigned int, int*, float *);
void calculate_pairwise_dists(float *, int, int, float *);
void find_pairwise_min(float *, int, float *, int *);
void merge_clusters(int *, int, int, int);
float calculate_dist(float *, int, int, int);
void print_float_matrix(float *, int, int);
void print_int_matrix(int *, int, int);
int get_parent(int, int *);

// Kernel functions
__global__ void calculate_pairwise_dists_cuda(float *, float *, unsigned int, unsigned int);
__global__ void find_pairwise_min_cuda(float * dist_matrix_d, int n, float* entry, int * indices, float* values);
__global__ void min_reduction(float *, float*, int);
__global__ void remove_cluster(float * dist_matrix_d, int right_cluster, int n);
__global__ void update_cluster(float * dist_matrix_d, int left_cluster, int right_cluster, int n);

/*************************** Helper Functions **************************************/
void print_float_matrix(float * a, int n, int m){
  for(int i=0; i<n; i++){
    for(int j=0; j<m; j++)
      printf("%f ", a[index(i, j, m)]);
    printf("\n");
  }
}

void print_int_matrix(int * a, int n, int m){
  for (int i=0; i<n; i++){
    for(int j=0; j<m; j++)
      printf("%d ", a[index(i,j,m)]);
    printf("\n");
  }
}

void load_data(float * dataset, int n, int m) {
  srand((unsigned int) 0);
  for (int i = 0; i < n; i ++) {
    for (int j = 0; j < m; j++) {
      // assign numbers between 0 and RANGE
      dataset[index(i, j, m)] = ((float)rand()/(float)(RAND_MAX)) * RANGE - RANGE/2.0;
    } 
  }
  if (PRINT_LOG){
    printf("Dataset:\n");
    print_float_matrix(dataset, n, m);
  }
}

void load_test_data(float * dataset) {
  float arr[6][2] = {
    {0.0,0.0},
    {1.0,1.0},
    {10.0,10.0},
    {11.0,11.0},
    {-100.0,-100.0},
    {-111.0,111.0}};

  int n = 6;
  int m = 2;

  for (int i = 0; i < n; i ++) {
    for (int j = 0; j < m; j++) {
      dataset[index(i, j, m)] = arr[i][j];
    } 
  }

  if (PRINT_LOG){
    printf("Dataset:\n");
    print_float_matrix(dataset, n, m);
  }
}


/**************************** main() *************************************/
int main(int argc, char * argv[])
{
  //Define variables
  //unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = atoi(argv[3]); // CPU or GPU
  int n = atoi(argv[1]);
  int m = atoi(argv[2]);

  printf("Hierarchical Clustering:\n");
  printf("Dataset size: %d x %d\n", n, m);
  printf("Device Type: %d\n", type_of_device);
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  // Validate
  /*if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }*/

  //Load data
  float * dataset;
  dataset = (float *)calloc(n*m, sizeof(float));
  if( !dataset )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", n, m);
   exit(1);
  }
  load_data(dataset, n, m);
  //load_test_data(dataset);
  printf("Data loaded!\n");
  
  type_of_device = atoi(argv[3]);

  float dendrogram[(n-1)*3];
  int * result;
  result = (int *)calloc(n, sizeof(int));
  if( type_of_device == 0 ) { 
    // The CPU sequential version 
    start = clock();
    seq_clustering(dataset, n, m, result, dendrogram);    
    end = clock();
  } else {
    // The GPU version
     start = clock();
     gpu_clustering(dataset, n, m, result, dendrogram);
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);

  free(dataset);
  free(result);

  return 0;

}


/*****************  The CPU sequential version **************/
void  seq_clustering(float * dataset, unsigned int n, unsigned int m, int* result, float * dendrogram)
{
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  if( !result ) {
   fprintf(stderr, " Cannot allocate result %u array\n", n);
   exit(1);
  }

  for (int i = 0; i < n; i++) result[i] = i;

  float* dist_matrix = (float *)calloc(n*n, sizeof(float));
  if( !dist_matrix ) {
   fprintf(stderr, " Cannot allocate dist_matrix %u array\n", n*n);
   exit(1);
  }

  // O(n*n*m) -> GPU
  start = clock();
  calculate_pairwise_dists(dataset, n, m, dist_matrix);
  if (PRINT_LOG)
    print_float_matrix(dist_matrix, n, n);
  end = clock();

  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  if (PRINT_ANALYSIS)
    printf("Time taken for distance computation: %lf\n", time_taken);
  
  start = clock();
  for (int iteration=0; iteration < n - 1; iteration++) {
    
    float entry[3]; 
    // O(I*n*n) -> GPU
    
    find_pairwise_min(dist_matrix, n, entry, result);
    
    
    dendrogram[index(iteration, 0, 3)] = entry[0];
    dendrogram[index(iteration, 1, 3)] = entry[1];
    dendrogram[index(iteration, 2, 3)] = entry[2];
    // O(I*n) -> amortized O(I)
    
    merge_clusters(result, (int)entry[0], (int)entry[1], n);
    
    
    if (PRINT_LOG){
      printf("Iteartion #%d\n", iteration);
      printf("Min Indices: %d, %d\n", (int)entry[0], (int)entry[1]);
      print_int_matrix(result, 1, n);
    }
    
  }

  end = clock();
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  if (PRINT_ANALYSIS)
      printf("Time taken for merge cluster, Iteration %lf\n",time_taken);
    
  for (int i=0; i<n; i++) result[i] = get_parent(i, result);

  if (PRINT_LOG){
    printf("Cluster IDs:\n");
    print_int_matrix(result, 1, n);
    printf("Dendrogram:\n");
    print_float_matrix(dendrogram, n-1, 3);
  }

  free(dist_matrix);
}

void calculate_pairwise_dists(float * dataset, int n, int m, float * dist_matrix) {
  // O(n)
  // for (int i = 0; i < n*n; i++) dist_matrix[i] = FLT_MAX;
  
  // O(n*n*m)
  for (int i = 0; i < n; i++) {
    for (int j = i+1; j < n; j++) {
      // O(m)
      dist_matrix[index(i, j, n)] = calculate_dist(dataset, i, j, m);
    }
  }  
}

// passing vec1_i and vec2_i instead of float * as dist_matrix is 1-D
float calculate_dist(float * dataset, int i, int j, int dim) {
  float dist = 0;
  // O(m)
  for (int mi = 0; mi < dim; mi++) {
    float x = (dataset[index(i, mi, dim)] - dataset[index(j,mi,dim)]);
    dist += x * x;
  }
  return dist;
}


int get_parent(int curr_parent, int* parents) {
  if (parents[curr_parent] == curr_parent) return curr_parent;
  parents[curr_parent] = get_parent(parents[curr_parent], parents);
  return parents[curr_parent];
  // return get_parent(parents[curr_parent], parents);
}


void find_pairwise_min(float * dist_matrix, int n, float* entry, int* parents) {
  entry[0] = 0;
  entry[1] = 0;
  entry[2] = FLT_MAX;
  for (int i = 0; i < n; i++) {
    for (int j = i+1; j < n; j++) {
      if (get_parent(i, parents) != get_parent(j, parents)) {
      // if (parents[i] != parents[j]) {
        float curr_dist = dist_matrix[index(i, j, n)];
        if (curr_dist < entry[2]) {
          entry[0] = i;
          entry[1] = j;
          entry[2] = curr_dist;
        }
      }
    }
  }
}


void merge_clusters(int * result, int data_point_i, int data_point_j, int dim) {
  if (!(data_point_i >= 0 && data_point_i < dim && data_point_j >= 0 && data_point_j < dim)) {
    printf("merge_clusters out of bounds");
    return;
  } 
  // int cluster_j = result[data_point_j];
  // for(int i=0; i<dim; i++)
  //   if(result[i] == cluster_j)
  //     result[i] = result[data_point_i];
  int parent_i = get_parent(data_point_i, result);
  result[get_parent(data_point_j, result)] = parent_i;
} 

/***************** The GPU version *********************/
/* This function can call one or more kernels if you want ********************/
void gpu_clustering(float * dataset, unsigned int n, unsigned int m, int * result, float * dendrogram){
  double time_taken;
  clock_t start, end;
  // int num_bytes = n*n*sizeof(float);
  for (int i = 0; i < n; i++) result[i] = i;

  // FIXME: Why we have dist_matrix in main memory? do we need it?
  float* dist_matrix = (float *)calloc(n*n, sizeof(float));
  if( !dist_matrix ) {
   fprintf(stderr, " Cannot allocate dist_matrix %u array\n", n*n);
   exit(1);
  }

  float * dist_matrix_d;
  hipMalloc((void**) &dist_matrix_d, n*n*sizeof(float));
  if (!dist_matrix_d) {
    fprintf(stderr, " Cannot allocate cuda dist_matrix %u array\n", n*n);
    exit(1);
  }

  float * dataset_d;
  hipMalloc((void**) &dataset_d, n*m*sizeof(float));
  if (!dataset_d) {
    fprintf(stderr, " Cannot allocate cuda dataset %u array\n", n*n);
    exit(1);
  }

  hipMemcpy(dist_matrix_d, dist_matrix, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dataset_d, dataset, n*m*sizeof(float), hipMemcpyHostToDevice);

  // Maximum number of threads per block in cuda1.cims.nyu.edu 
  int thread_cnt = 1024;
  int block_cnt = (int) ceil((float)n*n / thread_cnt);
  printf("Launching kernel with %d blocks and %d threads\n", block_cnt, thread_cnt);

  // O(1)
  start = clock();
  calculate_pairwise_dists_cuda<<<block_cnt, thread_cnt>>>(dataset_d, dist_matrix_d, n, m);
  hipDeviceSynchronize();
  hipMemcpy(dist_matrix, dist_matrix_d, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (PRINT_LOG) {
    printf("Dist Matrix:\n");
    print_float_matrix(dist_matrix, n, n);
  }
  end = clock();

  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  if (PRINT_ANALYSIS)
    printf("Time taken for distance computation: %lf\n", time_taken);
  
  start = clock();

  // Needs to be in shared memory
  int * indices;
  hipMalloc((void**) &indices, n*n*sizeof(int));
  float * values;
  hipMalloc((void**) &values, n*n*sizeof(float));

  // O(n)
  for (int iteration=0; iteration < n - 1; iteration++) {
    float entry[3]; 

    // O(log n)
    find_pairwise_min_cuda<<<block_cnt, thread_cnt>>> (dist_matrix_d, n, entry, indices, values);
    hipDeviceSynchronize();

    // Merge right cluster to left
    dendrogram[index(iteration, 0, 3)] = entry[0];
    dendrogram[index(iteration, 1, 3)] = entry[1];
    dendrogram[index(iteration, 2, 3)] = entry[2];

    // O(1)
    // Update left cluster's distance with all others
    update_cluster<<<block_cnt, thread_cnt>>> (dist_matrix_d, (int)entry[0], (int)entry[1], n);
    hipDeviceSynchronize();

    // Remove right clusters from further consideration
    remove_cluster<<<block_cnt, thread_cnt>>>(dist_matrix_d, (int)entry[1], n);
    hipDeviceSynchronize();
  
    if (PRINT_LOG){
      printf("Iteartion #%d\n", iteration);
      printf("Min Indices: %d, %d\n", (int)entry[0], (int)entry[1]);
      // print_int_matrix(result, 1, n);
    }
  }

  hipMemcpy(dist_matrix, dist_matrix_d, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (PRINT_LOG) {
    printf("Dist Matrix:\n");
    print_float_matrix(dist_matrix, n, n);
  }

  end = clock();
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  if (PRINT_ANALYSIS)
    printf("Time taken for merge cluster %lf\n", time_taken);
    
   if (PRINT_LOG){
  //  printf("Cluster IDs:\n");
  //   print_int_matrix(result, 1, n);
     printf("Dendrogram:\n");
     print_float_matrix(dendrogram, n-1, 3);
   }

  free(dist_matrix);
  hipFree(dataset_d);
  hipFree(dist_matrix_d);
}

/*
  Right is being merged to left
  So remove all distance entries for right with any other cluster 
*/ 
__global__ void update_cluster(float * dist_matrix_d, int left_cluster, int right_cluster, int n) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= n) return;

  int i = index/n;
  int j = index%n;
  if (i == left_cluster) {
    float new_min = min(dist_matrix_d[index(i, j, n)], dist_matrix_d[index(right_cluster, j, n)]);
    dist_matrix_d[index(i, j, n)] = new_min;
  } else if (j == left_cluster) {
    float new_min = min(dist_matrix_d[index(i, j, n)], dist_matrix_d[index(i, right_cluster, n)]);
    dist_matrix_d[index(i, j, n)] = new_min;
  }

  __syncthreads();
}

/*
  Right is being merged to left
  So remove all distance entries for right with any other cluster 
*/ 
__global__ void remove_cluster(float * dist_matrix_d, int right_cluster, int n) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= n) return;

  int i = index/n;
  int j = index%n;
  if (i == right_cluster || j == right_cluster) {
    dist_matrix_d[index] = FLT_MAX;
  }

  __syncthreads();
}

__global__ void calculate_pairwise_dists_cuda(float * dataset, float * dist_matrix, unsigned int n, unsigned int m)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  // Dont update if thread id is outside of the box
  if (index < n*n){
    int i = index / n;
    int j = index % n;
    if (i<n && j < n) {
      if (i == j) dist_matrix[index(i, j, n)] = FLT_MAX;
      else {
        float dist = 0;
        for(int mi=0; mi<m; mi++){
          float x = (dataset[index(i, mi, m)] - dataset[index(j,mi,m)]);
          dist += x * x;
        }
        dist_matrix[index(i, j, n)] = dist;
      }
    }
  }
}

__global__ void find_pairwise_min_cuda(float * dist_matrix_d, int n, float* entry, int * indices, float* values) {
  entry[0] = 0;
  entry[1] = 0;
  entry[2] = FLT_MAX;

  int index = threadIdx.x + blockIdx.x * blockDim.x;

  // indices and values needs to be shared
  // extern __shared__ int indices[];
  // extern __shared__ float values[];
  for (int stride = n*n/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (index < stride) {
      int left_idx = (stride == n*n/2) ? index : indices[index];
      int right_idx = (stride == n*n/2) ? index + stride : indices[index+stride];

      float left_val = dist_matrix_d[left_idx];
      // We can be outside of boundary in first iteration, handle it gracefully
      float right_val = FLT_MAX;
      if (right_idx < n*n) {
        right_val = dist_matrix_d[right_idx];
      }

      printf("find_pairwise_min_cuda - left_idx %d, left_val %.2f and right_idx %d, right_val %.2f | index %d, stride %d, n %d\n", 
      left_idx, left_val, right_idx, right_val, index, stride, n);

      if (left_val <= right_val) {
        indices[left_idx] = left_idx;
        values[left_idx] = left_val;
      } else {
        indices[left_idx] = right_idx;
        values[left_idx] = right_val;
      }
    }
  }

  __syncthreads();

  int min_val = values[0];
  int i = indices[0]/n;
  int j = indices[0]%n;

  // Always i should be smaller than j
  // That is cluster with higher index gets merged to the cluster with lower index
  if (i > j) {
    int temp = i;
    i = j;
    j = temp;
  } 

  entry[0] = i;
  entry[1] = j;
  entry[2] = min_val;
}

// This is a multi block parralell reduction
// reduce in block_mins after kernel finishes
__global__ void min_reduction(float *arr, float* block_mins, int n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int next = 1, left, right;
  n /= 2;
  while (n > 0){
    if (index < n){
      left = index * next * 2;
      right = left + next;
      if (arr[left] < arr[right]){
        arr[left] = arr[right];
      }
    }
    next *= 2;
    n /= 2;
  }
  __syncthreads();
  if (threadIdx.x == 0)
    block_mins[blockIdx.x] = arr[0];
}
